﻿
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <random>

constexpr int MAX_N = 10;
char errorString[256];

// global host memory arrays.
int* g_symbolsOut;
int* g_countsOut;
int* g_in;
int* g_decompressed;

// Device memory used in PARLE
int* d_totalRuns;
int* d_symbolsOut;
int* d_countsOut;
int* d_in;
int* d_backwardMask;
int* d_scannedBackwardMask;
int* d_compactedBackwardMask;
int numSMs;

void printArray(int* arr, int n) {
	for (int i = 0; i < n; ++i) {
		printf("%d, ", arr[i]);
	}
	printf("\n");
}

int* generateData() 
{
	std::random_device rd;     // only used once to initialise (seed) engine
	std::mt19937 rng(rd());    // random-number engine used (Mersenne-Twister in this case)
	std::uniform_int_distribution<int> uni(1, 100); // guaranteed unbiased

	for (size_t i = 0; i < MAX_N; ++i) {
		//g_in[i] = uni(rng);
		g_in[i] = 1;
	}

	return g_in;
}

int rleCpu(int *in, int n, int* symbolsOut, int* countsOut) {

	if (n == 0)
		return 0; // nothing to compress!

	int outIndex = 0;
	int symbol = in[0];
	int count = 1;

	for (int i = 1; i < n; ++i) {
		if (in[i] != symbol) {
			// run is over.
			// So output run.
			symbolsOut[outIndex] = symbol;
			countsOut[outIndex] = count;
			outIndex++;

			// and start new run:
			symbol = in[i];
			count = 1;
		}
		else {
			++count; // run is not over yet.
		}
	}

	// output last run.
	symbolsOut[outIndex] = symbol;
	countsOut[outIndex] = count;
	outIndex++;

	return outIndex;
}

__global__ void compactKernel(int* g_in, int* g_scannedBackwardMask, int* g_compactedBackwardMask, int* g_totalRuns, int n) {
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {

		if (i == (n - 1)) {
			g_compactedBackwardMask[g_scannedBackwardMask[i] + 0] = i + 1;
			*g_totalRuns = g_scannedBackwardMask[i];
		}

		if (i == 0) {
			g_compactedBackwardMask[0] = 0;
		}
		else if (g_scannedBackwardMask[i] != g_scannedBackwardMask[i - 1]) {
			g_compactedBackwardMask[g_scannedBackwardMask[i] - 1] = i;
		}
	}
}

__global__ void scatterKernel(int* g_compactedBackwardMask, int* g_totalRuns, int* g_in, int* g_symbolsOut, int* g_countsOut) {
	int n = *g_totalRuns;

	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
		int a = g_compactedBackwardMask[i];
		int b = g_compactedBackwardMask[i + 1];

		g_symbolsOut[i] = g_in[a];
		g_countsOut[i] = b - a;
	}
}

__global__ void maskKernel(int *g_in, int* g_backwardMask, int n) {
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
		printf("%d\n", i);
		printf("%d\n", g_in[i] != g_in[i - 1]);
		if (i == 0) {
			g_backwardMask[i] = 1;
			printf("-%d\n", __LINE__);
		} else {
			g_backwardMask[i] = (g_in[i] != g_in[i - 1]);
			printf("--%d\n", __LINE__);
		}
	}
}

//native scan
__global__ void scan(int *g_odata, int *g_idata, int n) {
	extern __shared__ float temp[]; // allocated on invocation    
	int thid = threadIdx.x;   
	int pout = 0, pin = 1;   
	// Load input into shared memory.    
	// This is exclusive scan, so shift right by one    
	// and set first element to 0   
	temp[pout*n + thid] = (thid > 0) ? g_idata[thid-1] : 0;   
	__syncthreads();   
	for (int offset = 1; offset < n; offset *= 2)   {     
		pout = 1 - pout; // swap double buffer indices     
		pin = 1 - pout;     
		if (thid >= offset)       
			temp[pout*n+thid] += temp[pin*n+thid - offset];     
		else       
			temp[pout*n+thid] = temp[pin*n+thid];     
		__syncthreads();   
	}   
	g_odata[thid] = temp[pout*n+thid]; // write output 
} 

// run parle on the GPU
void parleDevice(int *d_in, int n,
	int* d_symbolsOut,
	int* d_countsOut,
	int* d_totalRuns
) {
	int tmp[MAX_N];

	const int blocks = 32 * numSMs;
	maskKernel<<<blocks,256>>>(d_in, d_backwardMask, n);

	hipMemcpy(tmp, d_backwardMask, n * sizeof(int), hipMemcpyDeviceToHost);

	printArray(tmp, n);

	hipMemcpy(tmp, d_in, n * sizeof(int), hipMemcpyDeviceToHost);

	printArray(tmp, n);

	scan<<<blocks,256>>>(d_backwardMask, d_scannedBackwardMask, n);
	compactKernel<<<blocks,256>>>(d_in, d_scannedBackwardMask, d_compactedBackwardMask, d_totalRuns, n);
	scatterKernel<<<blocks,256>>>(d_compactedBackwardMask, d_totalRuns, d_in, d_symbolsOut, d_countsOut);
}

bool verifyCompression(
	int* original, int n,
	int* compressedSymbols, int* compressedCounts, int totalRuns) {

	// decompress.
	int j = 0;

	int sum = 0;
	for (int i = 0; i < totalRuns; ++i) {
		sum += compressedCounts[i];
	}

	if (sum != n) {
		sprintf(errorString, "Decompressed and original size not equal %d != %d\n", n, sum);

		for (int i = 0; i < totalRuns; ++i) {
			int symbol = compressedSymbols[i];
			int count = compressedCounts[i];

			printf("%d, %d\n", count, symbol);
		}
		return false;
	}

	for (int i = 0; i < totalRuns; ++i) {
		int symbol = compressedSymbols[i];
		int count = compressedCounts[i];

		for (int k = 0; k < count; ++k) {
			g_decompressed[j++] = symbol;
		}
	}

	// verify the compression.
	for (int i = 0; i < n; ++i) {
		if (original[i] != g_decompressed[i]) {

			sprintf(errorString, "Decompressed and original not equal at %d, %d != %d\n", i, original[i], g_decompressed[i]);
			return false;
		}
	}

	return true;
}

// On the CPU do preparation to run parle, launch PARLE on GPU, and then transfer the result data to the CPU. 
void parleHost(int *h_in, int n,
	int* h_symbolsOut,
	int* h_countsOut) 
{
	int h_totalRuns;

	
	// transer input data to device.
	d_in = generateData();
	hipMemcpy(d_in, h_in, n * sizeof(int), hipMemcpyHostToDevice);

	// RUN.    
	parleDevice(d_in, n, d_symbolsOut, d_countsOut, d_totalRuns);

	hipDeviceSynchronize();

	// transer result data to host.
	hipMemcpy(h_symbolsOut, d_symbolsOut, n * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(h_countsOut, d_countsOut, n * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&h_totalRuns, d_totalRuns, sizeof(int), hipMemcpyDeviceToHost);

	printf("n = %d\n", n);
	printf("Original Size  : %d\n", n);
	printf("Compressed Size: %d\n", h_totalRuns * 2);

	if (!verifyCompression(
		d_in, n,
		g_symbolsOut, g_countsOut, h_totalRuns)) {
		printf("Failed test %s\n", errorString);
		printArray(d_in, n);

		exit(1);
	}
	else {
		printf("passed test!\n\n");
	}


}

void printError(const char* msg, hipError_t err) 
{
	fprintf(stderr, "Error: %s, Desription: %s", msg, hipGetErrorString(err));
}


int main()
{
	hipError_t cudaStatus;

	cudaStatus = hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);

	if (cudaStatus != hipSuccess) {
		printError("hipDeviceGetAttribute failed!", cudaStatus);
		return 1;
	}

	cudaStatus = hipSetDevice(0);

	if (cudaStatus != hipSuccess) {
		printError("hipSetDevice failed!", cudaStatus);
		return 1;
	}

	// allocate resources on device. These arrays are used globally thoughouts the program.
	hipMalloc((void**)&d_backwardMask, MAX_N * sizeof(int));
	hipMalloc((void**)&d_scannedBackwardMask, MAX_N * sizeof(int));
	hipMalloc((void**)&d_compactedBackwardMask, (MAX_N + 1) * sizeof(int));
	hipMalloc((void**)&d_totalRuns, sizeof(int));
	hipMalloc((void**)&d_in, MAX_N * sizeof(int));
	hipMalloc((void**)&d_countsOut, MAX_N * sizeof(int));
	hipMalloc((void**)&d_symbolsOut, MAX_N * sizeof(int));

	// allocate resources on the host. 
	g_in = new int[MAX_N];
	g_decompressed = new int[MAX_N];
	g_symbolsOut = new int[MAX_N];
	g_countsOut = new int[MAX_N];
			
	parleHost(d_in, MAX_N, d_symbolsOut, d_countsOut);

	hipFree(d_backwardMask);
	hipFree(d_scannedBackwardMask);
	hipFree(d_compactedBackwardMask);
	hipFree(d_in);
	hipFree(d_countsOut);
	hipFree(d_symbolsOut);
	hipFree(d_totalRuns);

	hipDeviceReset();

	// free host memory.
	delete[] g_in;
	delete[] g_decompressed;

	delete[] g_symbolsOut;
	delete[] g_countsOut;

	return 0;
}

